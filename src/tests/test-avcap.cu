
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_HoriDownscale2X_Y210(ushort1* pSrc, int nSrcStep,
	ushort1* pDst, int nDstStep, int nWidth_4, int nHeight) {
	int x_4 = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x_4 < nWidth_4 && y < nHeight) {
		int x = x_4 * 4;
		int x_2 = x_4 * 2;
		int nSrcIdx = y * nSrcStep + x * 2;
		int nDstIdx = y * nDstStep + x_2 * 2;

		pDst[nDstIdx + 0] = pSrc[nSrcIdx + 0]; // Y
		pDst[nDstIdx + 1] = pSrc[nSrcIdx + 1]; // U
		pDst[nDstIdx + 2] = pSrc[nSrcIdx + 4]; // Y
		pDst[nDstIdx + 3] = pSrc[nSrcIdx + 7]; // V
	}
}

hipError_t zppiHoriDownscale2X_Y210(uchar1* pSrc, int nSrcStep, uchar1* pDst, int nDstStep, int nWidth, int nHeight) {
	static int BLOCK_W = 16;
	static int BLOCK_H = 16;

	int nWidth_4 = nWidth / 4;
	dim3 grid((nWidth_4 + BLOCK_W-1) / BLOCK_W, (nHeight + BLOCK_H-1) / BLOCK_H, 1);
	dim3 block(BLOCK_W, BLOCK_H, 1);

	kernel_HoriDownscale2X_Y210<<<grid, block>>>((ushort1*)pSrc, nSrcStep / 2,
		(ushort1*)pDst, nDstStep / 2, nWidth_4, nHeight);

	return hipDeviceSynchronize();
}
